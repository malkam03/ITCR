#include "hip/hip_runtime.h"
/* 
 * File:   saxpy.cu
 * Author: Malcolm Davis
 * Course: Computer Architecture II
 * Created on May 13, 2018
 * Simple SAXPY(Single-precision Alpha*X Plus Y) operation with CUDA
 *
 * Ussage: 
 * ./argv[0] for default parameters and random vectors or;
 * ./argv[0] <array size>
 */

#include <stdint.h>
#include <stdio.h>
#include <stdlib.h>
#include <time.h>
#include <math.h>
#include <unistd.h>
#include <omp.h>

#define FLOAT_RAND_MAX 10000
#define VECTOR_SIZE 100000
#define BLOCK_SIZE 16

/*
 *GPU KERNEL
 */
__global__
void saxpy_cuda(long n, float a, float *x, float *y)
{
  long i = blockIdx.x*blockDim.x + threadIdx.x;
  if (i < n) y[i] = a*x[i] + y[i];
}

void generateFloatVector(float *vector, long size);
void printIntVector(float *vector, long size);
void SAXPY(long size, float alpha, float *X, float *Y);
void copyVector(float *dest, float *src, long size);


/*
 * Main method, retrive command line options and run the saxpy
 */
int main(int argc, char const *argv[])
{     
    const int printVectors = getenv("PRINT_VECTORS") ? 1 : 0;
    
    srand(time(NULL));

    // If the vector size is inserted then use it if not then use the default 
    long size = argc > 1 && atol(argv[1]) > 0 ? atol(argv[1]) : VECTOR_SIZE;
    float alpha = ((float)rand()/(float)(RAND_MAX)) * FLOAT_RAND_MAX;

    // Allocate memory for the vectors
    float *H_Y, *H_YY, *H_X;
    hipHostMalloc((void **) &H_Y, sizeof(float)*size);
    hipHostMalloc((void **) &H_YY, sizeof(float)*size);
    hipHostMalloc((void **) &H_X, sizeof(float)*size);
    // Generate random vectors
    generateFloatVector(H_Y, size);
    generateFloatVector(H_X, size);
    copyVector(H_YY, H_Y, size);

    //Timing
    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    //Allocate memory for the vectors on device
    float* D_X, *D_Y;
    hipMalloc((void **) &D_X, sizeof(float)*size);
    hipMalloc((void **) &D_Y, sizeof(float)*size);

    // copy matrix A and B from host to device memory
    hipMemcpy(D_X, H_X, sizeof(float)*size, hipMemcpyHostToDevice);
    hipMemcpy(D_Y, H_Y, sizeof(float)*size, hipMemcpyHostToDevice);

    unsigned int grid_cols = (size + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, 1);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel
    saxpy_cuda<<<dimGrid, dimBlock>>>(size, alpha, D_X, D_Y);
    // Transefr results from device to host 
    hipMemcpy(H_Y, D_Y, sizeof(float)*size, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on saxpy of %ld on GPU: %f ms.\n\n", size, gpu_elapsed_time_ms);

    //CPU
    hipEventRecord(start, 0);
    SAXPY(size, alpha, H_X, H_YY);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on saxpy of %ld on CPU: %f ms.\n\n", size, cpu_elapsed_time_ms);

    hipFree(D_X);
    hipFree(D_Y);
    hipHostFree(H_Y);
    hipHostFree(H_YY);
    hipHostFree(H_X);
    return 0;
}

/*
 * SAXPY Function R = aX + Y
 * @param size the lenght of the vectors
 * @param alpha the const to scale the vector X
 * @param X a vector of floats
 * @param Y a vector of floats
 */
void SAXPY(long size, float alpha, float *X, float *Y)
{
    long i;
    #ifdef PARALLEL
    #pragma omp parallel for  private(i) shared(size, alpha, X, Y)
    #endif
    for(i = 0; i<size; i++){
        Y[i]=alpha*X[i] + Y[i];
    }
}

/*
 * Function that fills a vector of size "size" with random numbers
 * @param (INPUT)size the length of the vector
 * @param (OUTPUT)vector the place where the data will be stored.  
 */
void generateFloatVector(float *vector, long size)
{   
    long i;
    #ifdef PARALLEL
    #pragma omp parallel for  private(i) shared(size, vector)
    #endif
    for(i=0; i<size;i++){
        vector[i] = ((float)rand()/(float)(RAND_MAX)) * FLOAT_RAND_MAX;
    }
}

/*
 * Function that copy a vector
 * @param (INPUT)size the length of the vector
 * @param (OUTPUT)dest the place where the data will be stored. 
 * @param (OUTPUT)src the place where the data will be stored.  
 */
void copyVector(float *dest, float *src, long size)
{   
    long i;
    #ifdef PARALLEL
    #pragma omp parallel for  private(i) shared(size, vector)
    #endif
    for(i=0; i<size;i++){
        dest[i] = src[i];
    }
}

/*
 * Function that prints a vector on screen
 * @param (INPUT)size the length of the vector
 * @param (INPUT)vector the place where the data will be stored. 
 */
void printIntVector(float *vector, long size)
{   
    printf("[");
    for(long i=0; i<size;i++){
        printf(" %f ", vector[i]);
    }
    printf("]\n");
}
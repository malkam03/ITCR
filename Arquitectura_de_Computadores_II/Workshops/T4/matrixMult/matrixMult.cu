#include "hip/hip_runtime.h"
/* 
 * File:   matrixMult.cu
 * Author: Malcolm Davis
 * Course: Computer Architecture II
 * Created on May 9, 2018
 * Simple Matrix Multiplication with CUDA
 * Based on https://github.com/lzhengchun/matrix-cuda
 * Ussage: 
 * ./argv[0] for default parameters and random vectors or;
 * ./argv[0] <matrix 1 n size> <matrix 1 m size(and matrix 2 n size)> <matrix 2 m size>
 */
#include <stdio.h>
#include <stdlib.h>
#include <assert.h>

#define INT_RAND_MAX 10000
#define BLOCK_SIZE 16
#define MATRIX_SIZE_N 4
#define MATRIX_SIZE_M 4
#define MATRIX_SIZE_P 4

typedef struct intMatrix{
  int * data;
  long nrows;
  long ncols;
} intMatrix;

void generateMatrix(struct intMatrix* mat);
void printMatrix(struct intMatrix* mat);
void matMult(struct intMatrix* A, struct intMatrix* B, struct intMatrix* C);


/*
 *GPU KERNEL
 */
__global__
void cudaMult(int *a, int *b, int *c, int m, int n, int p)
{ 
    int row = blockIdx.y * blockDim.y + threadIdx.y; 
    int col = blockIdx.x * blockDim.x + threadIdx.x;
    int sum = 0;
    if( col < p && row < m) 
    {
        for(int i = 0; i < n; i++) 
        {
            sum += a[row * n + i] * b[i * p + col];
        }
        c[row * p + col] = sum;
    }
} 


/*
 * Main method, retrive command line options, create the threads
 */
int main(int argc, char const *argv[])
{
    const int printMatrixB = getenv("PRINT_MATRIX") ? 1 : 0;
    srand(time(NULL));
    // If the vector size is inserted then use it if not then use the default 
    long m = argc > 1 && atol(argv[1]) > 0 ? atol(argv[1]) : MATRIX_SIZE_M;
    long n = argc > 2 && atol(argv[2]) > 0 ? atol(argv[2]) : MATRIX_SIZE_N;
    long p = argc > 3 && atol(argv[3]) > 0 ? atol(argv[3]) : MATRIX_SIZE_P;

    // Allocate memory for the Matrix
    struct intMatrix H_A, H_B, H_C, H_CC;
    H_A.nrows = m;
    H_A.ncols = n;
    H_B.nrows = n;
    H_B.ncols = p;
    H_C.nrows = m;
    H_C.ncols = p;
    H_CC.nrows = m;
    H_CC.ncols = p;
    hipHostMalloc((void **) &H_A.data, sizeof(int)*m*n);
    hipHostMalloc((void **) &H_B.data, sizeof(int)*n*p);
    hipHostMalloc((void **) &H_C.data, sizeof(int)*m*p);
    hipHostMalloc((void **) &H_CC.data, sizeof(int)*m*p);
    
    // Generate random Matrix
    generateMatrix(&H_A);
    generateMatrix(&H_B);

    float gpu_elapsed_time_ms, cpu_elapsed_time_ms;

    // some events to count the execution time
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    // start to count execution time of GPU version
    hipEventRecord(start, 0);
    // Allocate memory space on the device 
    int* D_A, *D_B, *D_C;
    hipMalloc((void **) &D_A, sizeof(int)*m*n);
    hipMalloc((void **) &D_B, sizeof(int)*n*p);
    hipMalloc((void **) &D_C, sizeof(int)*m*p);

    // copy matrix A and B from host to device memory
    hipMemcpy(D_A, H_A.data, sizeof(int)*m*n, hipMemcpyHostToDevice);
    hipMemcpy(D_B, H_B.data, sizeof(int)*n*p, hipMemcpyHostToDevice);

    unsigned int grid_rows = (m + BLOCK_SIZE - 1) / BLOCK_SIZE;
    unsigned int grid_cols = (p + BLOCK_SIZE - 1) / BLOCK_SIZE;
    dim3 dimGrid(grid_cols, grid_rows);
    dim3 dimBlock(BLOCK_SIZE, BLOCK_SIZE);

    // Launch kernel
    cudaMult<<<dimGrid, dimBlock>>>(D_A, D_B, D_C, m, n, p);
    // Transefr results from device to host 
    hipMemcpy(H_C.data, D_C, sizeof(int)*m*p, hipMemcpyDeviceToHost);
    hipDeviceSynchronize();
    // time counting terminate
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);

    // compute time elapse on GPU computing
    hipEventElapsedTime(&gpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %ldx%ld . %ldx%ld on GPU: %f ms.\n\n", m, n, n, p, gpu_elapsed_time_ms);

    // start the CPU version
    hipEventRecord(start, 0);

    matMult(&H_A, &H_B, &H_CC);

    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&cpu_elapsed_time_ms, start, stop);
    printf("Time elapsed on matrix multiplication of %ldx%ld . %ldx%ld on CPU: %f ms.\n\n", m, n, n, p, cpu_elapsed_time_ms);

    // validate results computed by GPU
    int all_ok = 1;
    for (int i = 0; i < m; ++i)
    {
        for (int j = 0; j < p; ++j)
        {
            if(H_C.data[i*p + j] != H_CC.data[i*p + j])
            {
                all_ok = 0;
            }
        }
    }

    // roughly compute speedup
    if(all_ok)
    {
        printf("all results are correct!!!, speedup = %f\n", cpu_elapsed_time_ms / gpu_elapsed_time_ms);
    }
    else
    {
        printf("incorrect results\n");
    }

    // free memory
    hipFree(D_A);
    hipFree(D_B);
    hipFree(D_C);
    hipHostFree(H_A.data);
    hipHostFree(H_B.data);
    hipHostFree(H_C.data);
    hipHostFree(H_CC.data);
    return 0;
}


/*
 * matMult Function C = A*B
 * @param C the return matrix
 * @param A a matrix of ints
 * @param B a matrix of ints
 */
void matMult(struct intMatrix* A, struct intMatrix* B, struct intMatrix* C){
  long i, j, k;
  double sum = 0;
  #ifdef PARALLEL
  #pragma omp parallel for private(i,j,k, sum) shared(A, B, C)
  #endif
  for (i = 0; i < A->nrows; i++) {
      for (j = 0; j < B->ncols; j++) {
        for (k = 0; k < B->nrows; k++) {
          sum = sum + A->data[i*A->nrows+k]*B->data[k*B->nrows+j];
        }
        C->data[i*C->nrows+j] = sum;
        sum = 0;
      }
    }
}


/*
 * Function that fills a vector of size "size" with random numbers
 * @param (INPUT)size the length of the vector
 * @param (OUTPUT)vector the place where the data will be stored.  
 */
void generateMatrix(struct intMatrix* matrix){
  long i, j;
  #ifdef PARALLEL
  #pragma omp parallel for private(i, j) shared(matrix)
  #endif
  for(i=0; i < matrix->nrows; i++){
    for(j=0; j < matrix->ncols; j++){
      matrix->data[i*matrix->nrows+j] = (int16_t)ceil(((double)rand()/(double)(RAND_MAX)) * INT_RAND_MAX);
    }
  }
}

/*
 * Function that prints a vector on screen
 * @param (INPUT)size the length of the vector
 * @param (INPUT)vector the place where the data will be stored. 
 */
void printMatrix(struct intMatrix* matrix){
  for(long i=0; i < matrix->nrows; i++){
    printf("[");
    for(long j=0; j < matrix->ncols; j++){
      printf(" %hd ", matrix->data[i*matrix->nrows+j]);
    }
    printf("]\n");
  }
}